#include "hip/hip_runtime.h"
#include <iostream>

#include "defs.hpp"
#include "hll.cuh"
#include "boundary.cuh"
#include "reconstruct.cuh"
#include "eos.cuh"

__device__ void advect(Real hydro2[][1+2*NG], Real *hydro3, Real dt){
  //float dx = hydro[NHYDRO*(1+2*NG)+1]-hydro[NHYDRO*(1+2*NG)];
  Real dx = hydro2[NHYDRO][1]-hydro2[NHYDRO][0];
  //float u = hydro[1+2*NG+1];
  Real u = hydro2[1][1];
  
  if (u > 0.0){
    hydro3[0] = hydro2[0][1] - dt * u * (hydro2[0][1] - hydro2[0][0] ) / dx;
  }else{
    hydro3[0] = hydro2[0][1] - dt * u * (hydro2[0][2] - hydro2[0][1] ) / dx;
  }
  
}

__device__ void getflux (Real *hydro, Real *flx){
  
}

__global__ void kernel( Real *hydro, Real *hydro1, Real *x1c, Real *x1f , Real dt ){
  __shared__ Real hydrot[NHYDRO][MB + 2 * NG];
  __shared__ Real xct[MB + 2 * NG];
  __shared__ Real xft[MB + 2 * NG + 1];
  __shared__ Real faceL[MB+1][NHYDRO];
  __shared__ Real faceR[MB+1][NHYDRO];
  __shared__ Real flux[MB+1][NHYDRO];

  Real stencil[NHYDRO+2][2*HST];
  Real stencilB[NHYDRO+2][2*HST];
  Real out[NHYDRO];
  Real gamma = GAM;
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int idx_t = threadIdx.x + NG;
  //read input into shared memory
  
  for(int i=0; i<NHYDRO;++i){
    hydrot[i][idx_t] = hydro[i*(N+2*NG)+NG+idx];
    xct[idx_t]=x1c[idx];
    xft[idx_t]=x1f[idx];
    if ( threadIdx.x < NG ) {
      hydrot[i][idx_t-NG] = hydro[i*(N+2*NG)+idx];
      hydrot[i][idx_t + MB] = hydro[i*(N+2*NG)+NG+idx+MB];
      xct[idx_t-NG] = x1c[idx-NG];
      xct[idx_t + MB] = x1c[idx+MB];
      xft[idx_t-NG] = x1f[idx-NG];
      xft[idx_t + MB] = x1f[idx+MB];
    }
    if ( threadIdx.x == MB-1) {
      xft[idx_t + 1 + NG] = x1f[idx + NG + 1];
    }
  }
  __syncthreads();

  for(int j = -NG; j<NG; j++){
    for(int i = 0; i < NHYDRO; ++i){
      stencil[i][j+NG] = hydrot[i][idx_t + j];
    }
    stencil[NHYDRO][j+NG] = xct[idx_t + j];
    stencil[NHYDRO+1][j+NG] = xft[idx_t + j];
  }
  
  // extra stencil at outer face
  if ( threadIdx.x == MB -1 ) {

  for(int j = -NG; j<NG; j++){
    for(int i = 0; i < NHYDRO; ++i){
      stencilB[i][j+NG] = hydrot[i][idx_t + j + 1];
    }
    stencilB[NHYDRO][j+NG] = xct[idx_t + j + 1];
    stencilB[NHYDRO+1][j+NG] = xft[idx_t + j + 1];
  }

  }
  for(int i=0; i<NHYDRO;++i){
    donorcell(stencil[i], faceL[idx_t-NG][i], faceR[idx_t-NG][i], stencil[NHYDRO], stencil[NHYDRO+1]);
    if ( threadIdx.x == MB -1 ) {
      donorcell(stencilB[i], faceL[idx_t+1-NG][i], faceR[idx_t+1-NG][i],stencilB[NHYDRO], stencilB[NHYDRO+1]);
    }
  }

  __syncthreads();
   
  hll( faceL[idx_t-NG], faceR[idx_t-NG], flux[idx_t-NG], 1, gamma);
  if ( threadIdx.x == MB -1 ) {
    hll( faceL[idx_t + 1 -NG], faceR[idx_t + 1 -NG], flux[idx_t + 1 - NG], 1, gamma);
  }

  __syncthreads();

  // update with dt

  for (int i=0; i<NHYDRO;++i) {
    out[i] = hydrot[i][idx_t] + flux[idx_t - NG][i] * dt - flux[idx_t + 1 -NG][i] * dt;
  }
  for(int i=0; i<NHYDRO;++i) hydro1[i*(N+2*NG)+NG+idx] = out[i];
  
}

__global__ void kernel_copy(Real *in, Real *out){
  int idx =  threadIdx.x + blockIdx.x * blockDim.x;
  
  for(int i=0; i<NHYDRO;++i) {
    out[i*(N+2*NG)+NG+idx] = in[i*(N+2*NG)+NG+idx];
 //   if ( idx < NG ) {
 //   out[i*(N+2*NG)+idx] = in[i*(N+2*NG)+idx];
 //   out[i*(N+2*NG)+NG+idx + N] = in[i*(N+2*NG)+NG+idx+N];
  //  }
  }
}


void initial(Real *hydro, int ngrid){
  for(int k = 0; k < nx3 + 2*NG; ++k){
    x3c[k] = (k-NG)*0.1+0.05;
    x3f[k] = (k-NG)*0.1;
    for(int j = 0; j < nx2 + 2*NG; ++j){
      x2c[j] = (j-NG)*0.1+0.05;
      x2f[j] = (j-NG)*0.1;
      int loczy = j*(nx1+2*NG)+k*(nx2+2*NG)*(nx1+2*NG);
      for(int i = 0; i < nx1 + 2*NG; ++i){
      //hydro[NHYDRO*(N+2*NG)+i] = (i-NG)*0.1+0.05;
        x1c[i] = (i-NG)*0.1+0.05;
        x1f[i] = (i-NG)*0.1;
        if ( (i-NG)*0.1 < 50.){
            
	    hydro[loczy+i]=1.;
            hydro[IEN*ngrid+loczy+i] = 1.0;
	    hydro[IM1*ngrid+loczy+i] = 0.0;
	    hydro[IM2*ngrid+loczy+i] = 0.0;
	    hydro[IM3*ngrid+loczy+i] = 0.0;
         }
         else{
	    hydro[i]=0.125;
            hydro[IEN*ngrid+loczy+i] = 0.1;
	    hydro[IM1*ngrid+loczy+i] = 0.0;
	    hydro[IM2*ngrid+loczy+i] = 0.0;
	    hydro[IM3*ngrid+loczy+i] = 0.0;
         }
       }
     }
  }
}
int main(){

  Real *hydro, *hydro1, *x1c, *x1f;
  int ngrid = (nx3+2*NG)*(nx2+2*NG)*(nx1+2*NG);
  int nBytes = (nx3+2*NG)*(nx2+2*NG)*(nx1+2*NG)*(NHYDRO)*sizeof(Real);
  int nBytesx1c = (nx1+2*NG)*sizeof(Real);
  int nBytesx1f = (nx1+2*NG+1)*sizeof(Real);
  int nBytesx2c = (nx2+2*NG)*sizeof(Real);
  int nBytesx2f = (nx2+2*NG+1)*sizeof(Real);
  int nBytesx3c = (nx3+2*NG)*sizeof(Real);
  int nBytesx3f = (nx3+2*NG+1)*sizeof(Real);
  Real dt=0.03;

  hipMallocManaged(&hydro,nBytes);
  hipMallocManaged(&hydro1,nBytes);
  hipMallocManaged(&x1c,nBytesx1c);
  hipMallocManaged(&x1f,nBytesx1f);
  hipMallocManaged(&x2c,nBytesx2c);
  hipMallocManaged(&x2f,nBytesx2f);
  hipMallocManaged(&x3c,nBytesx3c);
  hipMallocManaged(&x3f,nBytesx3f);


  //initialize hydro
  initial(hydro,ngrid);
  initial(hydro1,ngrid);


  FILE *fp;
  fp = fopen("hydro.bin","wb");
  fwrite(hydro,sizeof(Real),nBytes,fp);
  fclose(fp);

  for (int step=0; step< nstep;step++){
    kernel<<<nx3*nx2*nx1/MB,MB>>>(hydro, hydro1, x1c, x1f, dt);
    hipDeviceSynchronize();
    openbc( hydro, hydro+(IM1*(N+2*NG)),hydro+(IEN*(N+2*NG)));         
    kernel_copy<<<N/MB,MB>>>(hydro1, hydro);
    hipDeviceSynchronize();
    if ( step % 1000 == 0 ) std::cout<<"step = "<<step<<std::endl;
  }

//  hipDeviceSynchronize();
  fp = fopen("hydro1.bin","wb");
  fwrite(hydro,sizeof(Real),nBytes,fp);
  fclose(fp);
  hipFree(hydro);
  hipFree(hydro1);
}
